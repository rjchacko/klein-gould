#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cutil.h>

#include "qcd.h"


#define REDUCE_THREADS 128
#define REDUCE_MAX_BLOCKS 64


__global__ void axpbyKernel(float a, float *x, float b, float *y, int len) {
    unsigned int i = blockIdx.x*(blockDim.x) + threadIdx.x;
    unsigned int gridSize = gridDim.x*blockDim.x;
    while (i < len) {
        y[i] = a*x[i] + b*y[i];
        i += gridSize;
    } 
}

// performs the operation y[i] = a*x[i] + b*y[i]
void axpbyCuda(float a, float *x, float b, float *y, int len) {
    int blocks = min(REDUCE_MAX_BLOCKS, max(len/REDUCE_THREADS, 1));
    dim3 dimBlock(REDUCE_THREADS, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    axpbyKernel<<<dimGrid, dimBlock>>>(a, x, b, y, len);
}


// performs the operation y[i] = a*x[i] + y[i]
void axpyCuda(float a, float *x, float *y, int len) {
    axpbyCuda(a, x, 1.0, y, len);
}

// performs the operation y[i] = x[i] + a*y[i]
void xpayCuda(float *x, float a, float *y, int len) {
    axpbyCuda(1.0, x, a, y, len);
}

// performs the operation y[i] -= x[i] (minus x plus y)
void mxpyCuda(float *x, float *y, int len) {
    axpbyCuda(-1.0, x, 1.0, y, len);
}


//
// float sumCuda(float* d_idata, int n) {}
//
#define REDUCE_FUNC_NAME(suffix) sum##suffix
#define REDUCE_TYPES float *a
#define REDUCE_PARAMS a
#define REDUCE_OPERATION(i) a[i]
#include "reduce_core.cu"
#undef REDUCE_FUNC_NAME
#undef REDUCE_TYPES
#undef REDUCE_PARAMS
#undef REDUCE_OPERATION

//
// float normCuda(float* d_idata, int n) {}
//
#define REDUCE_FUNC_NAME(suffix) norm##suffix
#define REDUCE_TYPES float *a
#define REDUCE_PARAMS a
#define REDUCE_OPERATION(i) (a[i]*a[i])
#include "reduce_core.cu"
#undef REDUCE_FUNC_NAME
#undef REDUCE_TYPES
#undef REDUCE_PARAMS
#undef REDUCE_OPERATION

//
// float dotProductCuda(float* d_idata, int n) {}
//
#define REDUCE_FUNC_NAME(suffix) dotProduct##suffix
#define REDUCE_TYPES float *a, float *b
#define REDUCE_PARAMS a, b
#define REDUCE_OPERATION(i) (a[i]*b[i])
#include "reduce_core.cu"
#undef REDUCE_FUNC_NAME
#undef REDUCE_TYPES
#undef REDUCE_PARAMS
#undef REDUCE_OPERATION




void blasTest(int argc, char **argv) {
    CUT_DEVICE_INIT(argc, argv);
    
    int n = 3*1<<8;
    float *h_data = (float *)malloc(n*sizeof(float));
    float *d_data;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_data,  n*sizeof(float)));
    
    double acc = 0;
    for (int i = 0; i < n; i++) {
        h_data[i] = i;
        acc += i*i;
    }
    CUDA_SAFE_CALL(hipMemcpy(d_data, h_data, n*sizeof(float), hipMemcpyHostToDevice));
    
    printf("Size: %f MiB\n", (float)n*sizeof(float) / (1 << 20));
    printf("cuda: %f, expected: %f\n", dotProductCuda(d_data, d_data, n), acc);
    
    CUDA_SAFE_CALL( hipFree(d_data) );
    free(h_data);
}

void axpbyTest(int argc, char **argv) {
    CUT_DEVICE_INIT(argc, argv);
    
    int n = 3 * 1 << 20;
    float *h_x = (float *)malloc(n*sizeof(float));
    float *h_y = (float *)malloc(n*sizeof(float));
    float *h_res = (float *)malloc(n*sizeof(float));
    
    float *d_x, *d_y;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_x,  n*sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_y,  n*sizeof(float)));
    
    for (int i = 0; i < n; i++) {
        h_x[i] = 1;
        h_y[i] = 2;
    }
    
    CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, n*sizeof(float), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, n*sizeof(float), hipMemcpyHostToDevice));
    
    axpbyCuda(4, d_x, 3, d_y, n/2);
    
    CUDA_SAFE_CALL( hipMemcpy( h_res, d_y, n*sizeof(float), hipMemcpyDeviceToHost) );

    for (int i = 0; i < n; i++) {
        float expect = (i < n/2) ? 4*h_x[i] + 3*h_y[i] : h_y[i];
        if (h_res[i] != expect)
            printf("FAILED %d : %f != %f\n", i, h_res[i], h_y[i]);
    }
    
    CUDA_SAFE_CALL( hipFree(d_y) );
    CUDA_SAFE_CALL( hipFree(d_x) );
    free(h_x);
    free(h_y);
}
