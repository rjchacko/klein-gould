#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cutil.h>
#include "qcd.h"

#define BLOCK_DIM (64) // threads per block
#define GRID_DIM (Nh/BLOCK_DIM) // there are Nh threads in total

#define SPINOR_SIZE (24) // spinors have 4*3*2 floats
#define PACKED_GAUGE_SIZE (4*20) // gauge matrices rounded up to fit float4 elements
#define SPINOR_BYTES (SPINOR_SIZE*sizeof(float))
#define PACKED_GAUGE_BYTES (PACKED_GAUGE_SIZE*sizeof(float))


// ----------------------------------------------------------------------
// Cuda code

float4 *d_gaugeEven, *d_gaugeOdd;
float4 *d_spinorEven, *d_spinorOdd;

texture<float4, 1, hipReadModeElementType> gauge0Tex;
texture<float4, 1, hipReadModeElementType> gauge1Tex;
texture<float4, 1, hipReadModeElementType> spinorTex;


__global__ void
dslashKernel(float4* g_out, int oddBit) {
    #include "dslash_core.cu"
}

__global__ void
dslashDaggerKernel(float4* g_out, int oddBit) {
    #include "dslash_dagger_core.cu"
}

// ----------------------------------------------------------------------

void packGaugeField(float4 *res, float **gauge) {
    for (int dir = 0; dir < 4; dir++) {
        for (int i = 0; i < Nh; i++) {
            for (int j = 0; j < 5; j++) {
                float a1, a2, a3=0, a4=0;
                a1 = gauge[dir][i*18 + j*4 + 0];
                a2 = gauge[dir][i*18 + j*4 + 1];
                if (j < 4) {
                    a3 = gauge[dir][i*18 + j*4 + 2];
                    a4 = gauge[dir][i*18 + j*4 + 3];
                }
                float4 f4 = {a1, a2, a3, a4};
                res[(dir*5+j)*Nh + i] = f4;
            }
        }
    }
}

void packSpinorField(float4 *res, float *spinor) {
    for (int i = 0; i < Nh; i++) {
        for (int j = 0; j < 6; j++) {
            float a1 = spinor[i*(6*4) + j*(4) + 0];
            float a2 = spinor[i*(6*4) + j*(4) + 1];
            float a3 = spinor[i*(6*4) + j*(4) + 2];
            float a4 = spinor[i*(6*4) + j*(4) + 3];
            float4 f4 = {a1, a2, a3, a4};
            res[j*Nh + i] = f4;
        }
    }
}

void unpackSpinorField(float *res, float4 *spinorPacked) {
    for (int i = 0; i < Nh; i++) {
        for (int j = 0; j < 6; j++) {
            float4 f4 = spinorPacked[j*Nh + i];
            res[i*(6*4) + j*(4) + 0] = f4.x;
            res[i*(6*4) + j*(4) + 1] = f4.y;
            res[i*(6*4) + j*(4) + 2] = f4.z;
            res[i*(6*4) + j*(4) + 3] = f4.w;
        }
    }
}

void sendGaugeField(float **gaugeEven, float **gaugeOdd) {
    float4 *packed1 = (float4*) malloc(Nh*PACKED_GAUGE_BYTES);
    float4 *packed2 = (float4*) malloc(Nh*PACKED_GAUGE_BYTES);
    packGaugeField(packed1, gaugeEven);
    packGaugeField(packed2, gaugeOdd);
    CUDA_SAFE_CALL(hipMemcpy(d_gaugeEven, packed1, Nh*PACKED_GAUGE_BYTES, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_gaugeOdd, packed2, Nh*PACKED_GAUGE_BYTES, hipMemcpyHostToDevice));
    free(packed1);
    free(packed2);
}

void sendSpinorFieldEven(float *spinorEven) {
    float4 *packed = (float4*) malloc(Nh*SPINOR_BYTES);
    packSpinorField(packed, spinorEven);
    CUDA_SAFE_CALL(hipMemcpy(d_spinorEven, packed, Nh*SPINOR_BYTES, hipMemcpyHostToDevice));
    free(packed);
}

void sendSpinorFieldOdd(float *spinorOdd) {
    float4 *packed = (float4*) malloc(Nh*SPINOR_BYTES);
    packSpinorField(packed, spinorOdd);
    CUDA_SAFE_CALL(hipMemcpy(d_spinorOdd, packed, Nh*SPINOR_BYTES, hipMemcpyHostToDevice));
    free(packed);
}

void retrieveSpinorFieldEven(float *res) {
    float4 *packed = (float4*) malloc(Nh*SPINOR_BYTES);
    CUDA_SAFE_CALL(hipMemcpy(packed, d_spinorEven, Nh*SPINOR_BYTES, hipMemcpyDeviceToHost));
    unpackSpinorField(res, packed);
    free(packed);
}

void retrieveSpinorFieldOdd(float *res) {
    float4 *packed = (float4*) malloc(Nh*SPINOR_BYTES);
    CUDA_SAFE_CALL(hipMemcpy(packed, d_spinorOdd, Nh*SPINOR_BYTES, hipMemcpyDeviceToHost));
    unpackSpinorField(res, packed);
    free(packed);
}

void initializeCuda(int argc, char** argv) {
    CUT_DEVICE_INIT(argc, argv);
    CUDA_SAFE_CALL(hipMalloc((void**) &d_gaugeEven, Nh*PACKED_GAUGE_BYTES));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_gaugeOdd, Nh*PACKED_GAUGE_BYTES));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_spinorEven, Nh*SPINOR_BYTES));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_spinorOdd, Nh*SPINOR_BYTES));

    printf("Spinors: %d\n", Nh);
    printf("Global kb: %f\n", 2*Nh*(PACKED_GAUGE_BYTES+SPINOR_BYTES)/1024.);
    printf("Shared kb: %fkB\n", SHARED_BYTES/1024.);
}

void releaseCuda() {
    CUDA_SAFE_CALL(hipFree(d_gaugeEven));
    CUDA_SAFE_CALL(hipFree(d_gaugeOdd));
    CUDA_SAFE_CALL(hipFree(d_spinorEven));
    CUDA_SAFE_CALL(hipFree(d_spinorOdd));
}

void dslashCuda(int oddBit, int daggerBit) {
    if (oddBit) {
        hipBindTexture(0 /*offset*/, gauge0Tex, d_gaugeOdd, Nh*PACKED_GAUGE_BYTES); 
        hipBindTexture(0 /*offset*/, gauge1Tex, d_gaugeEven, Nh*PACKED_GAUGE_BYTES); 
    }
    else {
        hipBindTexture(0 /*offset*/, gauge0Tex, d_gaugeEven, Nh*PACKED_GAUGE_BYTES); 
        hipBindTexture(0 /*offset*/, gauge1Tex, d_gaugeOdd, Nh*PACKED_GAUGE_BYTES); 
    }
    hipBindTexture(0 /*offset*/, spinorTex, d_spinorEven, Nh*SPINOR_BYTES); 

    dim3 gridDim(GRID_DIM, 1, 1);
    dim3 blockDim(BLOCK_DIM, 1, 1);
    
    if (!daggerBit) {
        dslashKernel <<<gridDim, blockDim, SHARED_BYTES>>> ((float4 *)d_spinorOdd, oddBit);
    }
    else {
        dslashDaggerKernel <<<gridDim, blockDim, SHARED_BYTES>>> ((float4 *)d_spinorOdd, oddBit);
    }
    
    CUT_CHECK_ERROR("Kernel execution failed");
    hipDeviceSynchronize();
}


void printSpinorHalfField(float *spinor) {
    printSpinor(&spinor[0*(4*3*2)]);
    printf("...\n");
    printSpinor(&spinor[(Nh-1)*(4*3*2)]);
    printf("\n");    
}

int main(int argc, char **argv) {
    initializeCuda(argc, argv);
    unsigned int timer = 0;
    cutCreateTimer(&timer);
    
    // construct input fields
    float *gaugeEven[4], *gaugeOdd[4];
    for (int dir = 0; dir < 4; dir++) {
        gaugeEven[dir] = (float*)malloc(Nh*3*3*2*sizeof(float));
        gaugeOdd[dir]  = (float*)malloc(Nh*3*3*2*sizeof(float));
    }
    float *spinorEven = (float*)malloc(Nh*4*3*2*sizeof(float));
    float *spinorOdd  = (float*)malloc(Nh*4*3*2*sizeof(float));
    float *spinorRef  = (float*)malloc(Nh*4*3*2*sizeof(float));

    // copy inputs from host to device
    printf("Randomizing fields\n");
    constructGaugeField(gaugeEven, gaugeOdd);
    constructSpinorField(spinorEven);
    sendGaugeField(gaugeEven, gaugeOdd);
    sendSpinorFieldEven(spinorEven);
    
    int ODD_BIT = 0;
    int DAGGER_BIT = 0;
    
    // execute kernel
    printf("Beginning kernel execution\n");
    cutStartTimer(timer);
    const int LOOPS = 100;
    for (int i = 0; i < LOOPS; i++) {
        dslashCuda(ODD_BIT, DAGGER_BIT);
    }
    cutStopTimer(timer);

    // print timing information
    float millisecs = cutGetTimerValue(timer)/LOOPS;
    float secs = millisecs / 1000.;
    printf("Elapsed time %fms\n", millisecs);
    printf("GFLOPS = %f\n", 1e-9*1320*Nh/secs);
    printf("GiB = %f\n\n", Nh*(8*7+4)*3*2*sizeof(float)/(secs*(1<<30)));

    // compare to dslash reference implementation
    retrieveSpinorFieldOdd(spinorOdd);
    dslashReference(spinorRef, gaugeEven, gaugeOdd, spinorEven, ODD_BIT, DAGGER_BIT);
    printf("Reference:\n");
    printSpinorHalfField(spinorRef);
    printf("\nCUDA:\n");
    printSpinorHalfField(spinorOdd);
    CUTBoolean res = cutComparefe(spinorOdd, spinorRef, Nh*4*3*2, 1e-4);
    printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");
    
    // release memory
    for (int dir = 0; dir < 4; dir++) {
        free(gaugeEven[dir]);
        free(gaugeOdd[dir]);
    }
    free(spinorEven);
    free(spinorOdd);
    free(spinorRef);
    cutDeleteTimer(timer);
    releaseCuda();
}
