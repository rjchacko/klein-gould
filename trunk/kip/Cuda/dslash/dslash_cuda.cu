#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cutil.h>
#include "qcd.h"

#define BLOCK_DIM (64) // threads per block
#define GRID_DIM (Nh/BLOCK_DIM) // there are Nh threads in total

#define SPINOR_SIZE (24) // spinors have 4*3*2 floats
#define PACKED_GAUGE_SIZE (4*20) // gauge matrices rounded up to fit float4 elements
#define SPINOR_BYTES (SPINOR_SIZE*sizeof(float))
#define PACKED_GAUGE_BYTES (PACKED_GAUGE_SIZE*sizeof(float))


// ----------------------------------------------------------------------
// Cuda code

texture<float4, 1, hipReadModeElementType> gauge0Tex;
texture<float4, 1, hipReadModeElementType> gauge1Tex;
texture<float4, 1, hipReadModeElementType> spinorTex;


__global__ void
dslashKernel(float4* g_out, int oddBit) {
    #include "dslash_core.cu"
}

__global__ void
dslashDaggerKernel(float4* g_out, int oddBit) {
    #include "dslash_dagger_core.cu"
}


// ----------------------------------------------------------------------

void packGaugeField(float4 *res, float **gauge, int oddBit) {
    for (int dir = 0; dir < 4; dir++) {
        float *g = gauge[dir] + oddBit*(Nh*3*3*2);
        for (int i = 0; i < Nh; i++) {
            for (int j = 0; j < 5; j++) {
                float a1, a2, a3=0, a4=0;
                a1 = g[i*18 + j*4 + 0];
                a2 = g[i*18 + j*4 + 1];
                if (j < 4) {
                    a3 = g[i*18 + j*4 + 2];
                    a4 = g[i*18 + j*4 + 3];
                }
                float4 f4 = {a1, a2, a3, a4};
                res[(dir*5+j)*Nh + i] = f4;
            }
        }
    }
}

void packParitySpinor(float4 *res, float *spinor) {
    for (int i = 0; i < Nh; i++) {
        for (int j = 0; j < 6; j++) {
            float a1 = spinor[i*(6*4) + j*(4) + 0];
            float a2 = spinor[i*(6*4) + j*(4) + 1];
            float a3 = spinor[i*(6*4) + j*(4) + 2];
            float a4 = spinor[i*(6*4) + j*(4) + 3];
            float4 f4 = {a1, a2, a3, a4};
            res[j*Nh + i] = f4;
        }
    }
}

void unpackParitySpinor(float *res, float4 *spinorPacked) {
    for (int i = 0; i < Nh; i++) {
        for (int j = 0; j < 6; j++) {
            float4 f4 = spinorPacked[j*Nh + i];
            res[i*(6*4) + j*(4) + 0] = f4.x;
            res[i*(6*4) + j*(4) + 1] = f4.y;
            res[i*(6*4) + j*(4) + 2] = f4.z;
            res[i*(6*4) + j*(4) + 3] = f4.w;
        }
    }
}

CudaPSpinor allocateParitySpinor() {
    CudaPSpinor ret;
    CUDA_SAFE_CALL(hipMalloc((void**)&ret, Nh*SPINOR_BYTES));
    return ret;
}

CudaFullGauge loadGaugeField(float **gauge) {
    CudaFullGauge ret;
    CUDA_SAFE_CALL(hipMalloc((void **)&ret.even, Nh*PACKED_GAUGE_BYTES));
    CUDA_SAFE_CALL(hipMalloc((void **)&ret.odd,  Nh*PACKED_GAUGE_BYTES));

    float4 *packedEven = (float4*) malloc(Nh*PACKED_GAUGE_BYTES);
    float4 *packedOdd  = (float4*) malloc(Nh*PACKED_GAUGE_BYTES);
    packGaugeField(packedEven, gauge, 0);
    packGaugeField(packedOdd,  gauge, 1);
    CUDA_SAFE_CALL(hipMemcpy(ret.even, packedEven, Nh*PACKED_GAUGE_BYTES, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(ret.odd,  packedOdd,  Nh*PACKED_GAUGE_BYTES, hipMemcpyHostToDevice));    
    free(packedEven);
    free(packedOdd);
    
    return ret;
}

CudaPSpinor loadParitySpinor(float *spinor) {
    CudaPSpinor ret;
    CUDA_SAFE_CALL(hipMalloc((void**)&ret, Nh*SPINOR_BYTES));
    
    float4 *packed = (float4*) malloc(Nh*SPINOR_BYTES);
    packParitySpinor(packed, spinor);
    CUDA_SAFE_CALL(hipMemcpy(ret, packed, Nh*SPINOR_BYTES, hipMemcpyHostToDevice));
    free(packed);
    
    return ret;
}

CudaFullSpinor loadSpinorField(float *spinor) {
    CudaFullSpinor ret;
    ret.even = loadParitySpinor(spinor);
    ret.odd  = loadParitySpinor(spinor + Nh*SPINOR_BYTES);
    
    /*
    CUDA_SAFE_CALL(hipMalloc((void**)&ret.even, Nh*SPINOR_BYTES));
    CUDA_SAFE_CALL(hipMalloc((void**)&ret.odd,  Nh*SPINOR_BYTES));
    
    float4 *packedEven = (float4*) malloc(Nh*SPINOR_BYTES);
    float4 *packedOdd  = (float4*) malloc(Nh*SPINOR_BYTES);
    packSpinorField(packedEven, spinor, 0);
    packSpinorField(packedOdd,  spinor, 1);
    CUDA_SAFE_CALL(hipMemcpy(ret.even, packedEven, Nh*SPINOR_BYTES, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(ret.odd,  packedOdd,  Nh*SPINOR_BYTES, hipMemcpyHostToDevice));    
    free(packedEven);
    free(packedOdd);
    */
    
    return ret;
}

void freeParitySpinor(CudaPSpinor spinor) {
    CUDA_SAFE_CALL(hipFree(spinor));
}

void freeGaugeField(CudaFullGauge gauge) {
    CUDA_SAFE_CALL(hipFree(gauge.even));
    CUDA_SAFE_CALL(hipFree(gauge.odd));
}

void freeSpinorField(CudaFullSpinor spinor) {
    CUDA_SAFE_CALL(hipFree(spinor.even));
    CUDA_SAFE_CALL(hipFree(spinor.odd));
}

void retrieveParitySpinor(float *res, CudaPSpinor spinor) {
    float4 *packed = (float4*) malloc(Nh*SPINOR_BYTES);
    CUDA_SAFE_CALL(hipMemcpy(packed,  spinor,  Nh*SPINOR_BYTES, hipMemcpyDeviceToHost));
    unpackParitySpinor(res, packed);
    free(packed);
}

void retrieveSpinorField(float *res, CudaFullSpinor spinor) {
    retrieveParitySpinor(res, spinor.even);
    retrieveParitySpinor(res+Nh*SPINOR_BYTES, spinor.odd);
    
/*    float4 *packedEven = (float4*) malloc(Nh*SPINOR_BYTES);
    float4 *packedOdd = (float4*) malloc(Nh*SPINOR_BYTES);
    CUDA_SAFE_CALL(hipMemcpy(packedEven, spinor.even, Nh*SPINOR_BYTES, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(packedOdd,  spinor.odd,  Nh*SPINOR_BYTES, hipMemcpyDeviceToHost));
    unpackSpinorField(res, packedEven, 0);
    unpackSpinorField(res, packedEven, 1);
    free(packedEven);
    free(packedOdd);
    */
}

void dslashCuda(CudaPSpinor res, CudaFullGauge gauge, CudaPSpinor spinor, int oddBit, int daggerBit) {
    if (oddBit) {
        hipBindTexture(0 /*offset*/, gauge0Tex, gauge.odd, Nh*PACKED_GAUGE_BYTES); 
        hipBindTexture(0 /*offset*/, gauge1Tex, gauge.even, Nh*PACKED_GAUGE_BYTES); 
    }
    else {
        hipBindTexture(0 /*offset*/, gauge0Tex, gauge.even, Nh*PACKED_GAUGE_BYTES); 
        hipBindTexture(0 /*offset*/, gauge1Tex, gauge.odd, Nh*PACKED_GAUGE_BYTES); 
    }
    hipBindTexture(0 /*offset*/, spinorTex, spinor, Nh*SPINOR_BYTES); 

    dim3 gridDim(GRID_DIM, 1, 1);
    dim3 blockDim(BLOCK_DIM, 1, 1);
    
    if (!daggerBit) {
        dslashKernel <<<gridDim, blockDim, SHARED_BYTES>>> ((float4 *)res, oddBit);
    }
    else {
        dslashDaggerKernel <<<gridDim, blockDim, SHARED_BYTES>>> ((float4 *)res, oddBit);
    }
    
    CUT_CHECK_ERROR("Kernel execution failed");
    hipDeviceSynchronize();
}

int dslashCudaSharedBytes() {
    return SHARED_BYTES;
}


// Apply the even-odd preconditioned Dirac operator
void MatPCCuda(CudaPSpinor outEven, CudaFullGauge gauge, CudaPSpinor inEven, float kappa, CudaPSpinor tmp) {
    // full dslash operator
    dslashCuda(tmp, gauge, inEven, 1, 0);
    dslashCuda(outEven, gauge, tmp, 0, 0);
    
    // lastly apply the kappa term
    float kappa2 = -kappa*kappa;
    xpayCuda((float *)inEven, kappa2, (float *)outEven, Nh*spinorSiteSize);
}

// Apply the even-odd preconditioned Dirac operator
void MatPCDagCuda(CudaPSpinor outEven, CudaFullGauge gauge, CudaPSpinor inEven, float kappa, CudaPSpinor tmp) {
    // full dslash operator
    dslashCuda(tmp, gauge, inEven, 1, 1);
    dslashCuda(outEven, gauge, tmp, 0, 1);
    
    float kappa2 = -kappa*kappa;
    xpayCuda((float *)inEven, kappa2, (float *)outEven, Nh*spinorSiteSize);
}

void MatPCDagMatPCCuda(CudaPSpinor outEven, CudaFullGauge gauge, CudaPSpinor inEven, float kappa, CudaPSpinor tmp1, CudaPSpinor tmp2) {
    MatPCCuda(tmp2, gauge, inEven, kappa, tmp1);
    MatPCDagCuda(outEven, gauge, tmp2, kappa, tmp1);
}
