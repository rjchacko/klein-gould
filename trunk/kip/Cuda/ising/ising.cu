#include <math.h>
#include "ising.h"

NVIsing nvAllocate(int len, int dim) {
    NVIsing ret;
    ret.len = len;
    ret.dim = dim;
    ret.n = (int)powl(len, dim);
    hipMalloc((void**)&(ret.spins), (ret.n/32)*sizeof(unsigned int));
    return ret;
}

void nvFree(NVIsing ising) {
    hipFree(ising.spins);
}

void nvLoadSpins(NVIsing ising, unsigned int *spins) {
    hipMemcpy(ising.spins, spins, (ising.n/32)*sizeof(unsigned int), hipMemcpyHostToDevice);
}

void nvRetrieveSpins(NVIsing ising, unsigned int *spins) {
    hipMemcpy(spins, ising.spins, (ising.n/32)*sizeof(unsigned int), hipMemcpyDeviceToHost);
}





void nvUpdate(NVIsing ising, int parityTarget) {
    
}
