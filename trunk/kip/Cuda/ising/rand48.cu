
#include <hip/hip_runtime.h>

struct Rand48 {
    // strided iteration constants (48-bit, distributed on 2x 24-bit)
    uint2 A, C;
    // CUDA array -- random numbers for all threads
    uint2 *state;
    // random number for a single thread (used by CUDA device functions only)
    uint2 state0;
    
    // magic constants for rand48
    static const unsigned long long a = 0x5DEECE66DLL, c = 0xB;
    
    void init(int nThreads, int seed) {
        uint2* seeds = new uint2[ nThreads ];
        
        hipMalloc((void**) &state, sizeof(uint2)*nThreads);
        
        // calculate strided iteration constants
        unsigned long long A, C;
        A = 1LL; C = 0LL;
        for (unsigned int i = 0; i < nThreads; ++i) {
            C += A*c;
            A *= a;
        }
        this->A.x = A & 0xFFFFFFLL;
        this->A.y = (A >> 24) & 0xFFFFFFLL;
        this->C.x = C & 0xFFFFFFLL;
        this->C.y = (C >> 24) & 0xFFFFFFLL;
        
        // prepare first nThreads random numbers from seed
        unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
        for (unsigned int i = 0; i < nThreads; ++i) {
            x = a*x + c;
            seeds[i].x = x & 0xFFFFFFLL;
            seeds[i].y = (x >> 24) & 0xFFFFFFLL;
        }
        
        hipMemcpy(state, seeds, sizeof(uint2)*nThreads, hipMemcpyHostToDevice);
        
        delete[] seeds;
    }
    
    void destroy() {
        hipFree((void*) state);
    }
};

__device__ inline void rand48_loadState(Rand48 &r) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    r.state0 = r.state[i];
}

__device__ inline void rand48_storeState(Rand48 &r) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    r.state[i] = r.state0;
}

__device__ inline void rand48_iterate(Rand48 &r) {
    // state0 is 2x 24bit to handle overflows optimally, i.e.
    // in one operation.
    
    // the multiplication commands however give the low and hi 32 bit,
    // which have to be converted as follows:
    // 48bit in bytes = ABCD EF (space marks 32bit boundary)
    // R0             = ABC
    // R1             =    D EF
    
    unsigned int R0, R1;
    
    // low 24-bit multiplication
    const unsigned int lo00 = __umul24(r.state0.x, r.A.x);
    const unsigned int hi00 = __umulhi(r.state0.x, r.A.x);
    
    // 24bit distribution of 32bit multiplication results
    R0 = (lo00 & 0xFFFFFF);
    R1 = (lo00 >> 24) | (hi00 << 8);
    
    R0 += r.C.x; R1 += r.C.y;
    
    // transfer overflows
    R1 += (R0 >> 24);
    R0 &= 0xFFFFFF;
    
    // cross-terms, low/hi 24-bit multiplication
    R1 += __umul24(r.state0.y, r.A.x);
    R1 += __umul24(r.state0.x, r.A.y);
    
    R1 &= 0xFFFFFF;
    
    r.state0 = make_uint2(R0, R1);
}

__device__ inline int rand48_nextInt(Rand48 &r) {
    // get upper 31 (!) bits of the 2x 24bits
    int res = ( r.state0.x >> 17 ) | ( r.state0.y << 7 );
    rand48_iterate(r);
    return res;
}

// returns a float in the range (0, 1]
__device__ inline int rand48_nextFloat(Rand48 &r) {
    return ((float)rand48_nextInt(r)+1.0f) / (1<<31);
}
